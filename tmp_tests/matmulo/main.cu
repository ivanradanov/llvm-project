
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <random>
#include <chrono>
#include <assert.h>
#include <stdlib.h>
#include <cstdlib>
#include <omp.h>

int NITERATIONS = 1;

#ifndef BLOCK_SIZE
#define BLOCK_SIZE 32
#endif

__global__ void mat_mul(float *A, float *B, float *C, int size_x, int size_y)
{
	// these point to the first element of the first block we are considering
	float *a = A + blockIdx.y * BLOCK_SIZE * size_x;
	float *b = B + blockIdx.x * BLOCK_SIZE;
	float *c = C + blockIdx.y * BLOCK_SIZE * size_y + blockIdx.x * BLOCK_SIZE;


	int numBlocks = size_x / BLOCK_SIZE;

	float res = 0;

	for (int i = 0; i < numBlocks; a += BLOCK_SIZE, b += BLOCK_SIZE * size_y, i++) {

		// now a and b point to the first element of the block we are considering

		__shared__ float sa[BLOCK_SIZE][BLOCK_SIZE];
		__shared__ float sb[BLOCK_SIZE][BLOCK_SIZE];

		sa[threadIdx.y][threadIdx.x] = a[size_x * threadIdx.y + threadIdx.x];
		sb[threadIdx.y][threadIdx.x] = b[size_y * threadIdx.y + threadIdx.x];

		__syncthreads();


#pragma unroll
		for (int j = 0; j < BLOCK_SIZE; j++) {
			res += sa[threadIdx.y][j] * sb[j][threadIdx.x];
		}
		__syncthreads();

	}
	c[threadIdx.x + threadIdx.y * size_y] = res;
}

constexpr std::size_t default_alignment = sizeof(double) * 16;

inline void* aligned_malloc(size_t align, size_t size)
{
	assert(align >= sizeof(void*));

	if (size == 0) {
		return nullptr;
	}

	void* result = nullptr;
	int err = posix_memalign(&result, align, size);

	if (err != 0) {
		return nullptr;
	}

	return result;
}

void populate_array(float *a, int size) {
	for (int i = 0; i < size; ++i)
		a[i] = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
}

bool array_equal(float *a, float *b, int size) {
	double e = 1.e-6;  // epsilon
	for (int i = 0; i < size; ++i)
		if (fabs(a[i] - b[i]) > e)
			return false;
	return true;
}

void cpu_mat_mul(float *A, float *B, float *C, int size_m, int size_n, int size_k) {
  for (int m = 0; m < size_m; m++) {
    for (int n = 0; n < size_n; n++) {
	    int Ci = n + m * size_n;
	    C[Ci] = 0;
      for (int k = 0; k < size_k; k++) {
	      int Ai = k + m * size_k;
	      int Bi = n + k * size_n;
	      C[Ci] += A[Ai] * B[Bi];
      }
    }
  }
}

void print_mat(float *A, int size_m, int size_n) {
	for (int m = 0; m < size_m; m++) {
		for (int n = 0; n < size_n; n++) {
			int Ai = n + m * size_n;
			std::cout << std::fixed << std::setprecision(2) << A[Ai] << " ";
		}
		std::cout << std::endl;
	}
	std::cout << std::endl;
}

void run(int block_size, dim3 dimsA, dim3 dimsB) {

	std::cout << "dimsA "
	          << dimsA.x << " "
	          << dimsA.y << " "
	          << dimsA.z << std::endl;
	std::cout << "dimsB "
	          << dimsB.x << " "
	          << dimsB.y << " "
	          << dimsB.z << std::endl;

	float *A = (float *) aligned_malloc(default_alignment, sizeof(float) * dimsA.x * dimsA.y);
	float *B = (float *) aligned_malloc(default_alignment, sizeof(float) * dimsB.x * dimsB.y);

	// Random floats 0.0 - 1.0
	populate_array(A, dimsA.x * dimsA.y);
	populate_array(B, dimsB.x * dimsB.y);

	dim3 dimsC(dimsB.x, dimsA.y, 1);
	float *C = (float *) aligned_malloc(default_alignment, sizeof(float) * dimsC.x * dimsC.y);

	dim3 block(block_size, block_size);
  dim3 grid(dimsB.x / block.x, dimsA.y / block.y);

  std::cout << "Executing warmup" << std::endl;

  // warmup
  {
    for (int i = 0; i < NITERATIONS; ++i) {
      mat_mul<<<grid, block>>>(A, B, C, dimsA.x, dimsB.x);
    }
  }

  std::cout << "Executing kernel" << std::endl;

  auto start = std::chrono::high_resolution_clock::now();

  for (int i = 0; i < NITERATIONS; ++i) {
	  mat_mul<<<grid, block>>>(A, B, C, dimsA.x, dimsB.x);
  }

  auto end = std::chrono::high_resolution_clock::now();

  using namespace std::literals;
  auto us = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();
  double ms = us / 1000.0;
  double s = ms / 1000.0;
  std::cout << "Executed " << NITERATIONS << " iterations in " << us << "??s ??? " << ms << "ms ??? " << s << "s.\n";


  double matrix_flops = 2.0 * (double) dimsA.x * (double) dimsA.y * (double) dimsB.x;
  double giga = (double) 1000.0 * 1000.0 * 1000.0;
  double gflops = (NITERATIONS * matrix_flops / giga)  / s;

  std::cout << "GFlop/s: " << gflops << std::endl << std::endl;

  std::cout
	  << omp_get_max_threads() << ", "
	  << dimsA.y << ", "
	  << dimsB.x << ", "
	  << dimsA.x << ", "
	  << NITERATIONS << ", "
	  << ms << ", "
	  << gflops << ", "
	  << std::endl
    << std::endl;


  std::cout << "Running verification..." << std::endl;

  float *C2 = (float *) aligned_malloc(default_alignment, sizeof(float) * dimsC.x * dimsC.y);

  assert(dimsB.y == dimsA.x);

  start = std::chrono::high_resolution_clock::now();
  cpu_mat_mul(A, B, C2, dimsA.y, dimsB.x, dimsA.x);
  end = std::chrono::high_resolution_clock::now();

  us = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();
  ms = us / 1000.0;
  s = ms / 1000.0;
  std::cout << "Verification mat mul completed in  " << us << "??s ??? " << ms << "ms ??? " << s << "s.\n";

  auto success = array_equal(C, C2, dimsC.x * dimsC.y);

  if (success)
	  std::cout << "PASS" << std::endl;
  else
    std::cout << "FAILED" << std::endl;


  /*
    std::cout << "A" << std::endl;
    print_mat(A, dimsA.x, dimsA.y);
    std::cout << "B" << std::endl;
    print_mat(B, dimsB.x, dimsB.y);
    std::cout << "C" << std::endl;
    print_mat(C, dimsC.x, dimsC.y);
    std::cout << "C2" << std::endl;
    print_mat(C2, dimsC.x, dimsC.y);
  */


}

void test_cpu_mat_mul();

int main(int argc, char **argv) {

	int block_size = BLOCK_SIZE;

	if (argc != 1 && argc != 5 && argc != 2) {
		std::cout << "Usage: ./a.out <m> <n> <k> <n_iters>" << std::endl;
		return 1;
	}
	int m, n, k;
	if (argc == 5) {
    int i = 1;
    m = atoi(argv[i++]);
    n = atoi(argv[i++]);
    k = atoi(argv[i++]);
    assert(m % block_size == 0);
    assert(n % block_size == 0);
    assert(k % block_size == 0);
    NITERATIONS = atoi(argv[i++]);
	} else {
		m = 2 * block_size;
		n = 3 * block_size;
		k = 4 * block_size;
	}
	if (argc == 2) {
		test_cpu_mat_mul();
		return 1;
	}

	dim3 dimsA(k, m, 1);
	dim3 dimsB(n, k, 1);

	run(block_size, dimsA, dimsB);
	return 0;

}

void test_cpu_mat_mul() {
	float *A = (float *) aligned_malloc(default_alignment, sizeof(float) * 1000);
	float *B = (float *) aligned_malloc(default_alignment, sizeof(float) * 1000);
	float *C = (float *) aligned_malloc(default_alignment, sizeof(float) * 1000);
	int i = 0;
	A[i++] = 1;
	A[i++] = 2;
	A[i++] = 3;
	A[i++] = 4;
	A[i++] = 5;
	A[i++] = 6;
	A[i++] = 4;
	A[i++] = 5;
	A[i++] = 6;
	A[i++] = 4;
	A[i++] = 5;
	A[i++] = 6;

	i = 0;
	B[i++] = 7;
	B[i++] = 8;
	B[i++] = 9;
	B[i++] = 10;
	B[i++] = 11;
	B[i++] = 12;

	cpu_mat_mul(A, B, C, 4, 2, 3);

	std::cout << "A" << std::endl;
	print_mat(A, 4, 3);
	std::cout << "B" << std::endl;
	print_mat(B, 3, 2);
	std::cout << "C" << std::endl;
	print_mat(C, 4, 2);

	return;
}
